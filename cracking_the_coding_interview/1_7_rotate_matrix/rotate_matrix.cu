#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>
#include <tuple>
#include <vector>

__device__ __forceinline__ int idx(int x, int y, int width) {
  return y * width + x;
}

__global__ void rotate_matrix_dev(int* matrix, int n) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < (n + 1) / 2 && y < (n + 1) / 2) {
    // save top left
    int tmp = matrix[idx(x, y, n)];
    // replace top left with bottom left
    matrix[idx(x, y, n)] = matrix[idx(y, n - 1 - x, n)];
    // replace bottom left with bottom right
    matrix[idx(y, n - 1 - x, n)] = matrix[idx(n - 1 - x, n - 1 - y, n)];
    // replace bottom right with top right
    matrix[idx(n - 1 - x, n - 1 - y, n)] = matrix[idx(n - 1 - y, x, n)];
    // replace top right with temporary (top left)
    matrix[idx(n - 1 - y, x, n)] = tmp;
  }
}

void rotate_matrix(int* matrix, int n) {
  int* d_matrix;

  hipMalloc(&d_matrix, n * n * sizeof(int));
  hipMemcpy(d_matrix, matrix, n * n * sizeof(int), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(1, 1);
  dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

  rotate_matrix_dev<<<blocksPerGrid, threadsPerBlock>>>(d_matrix, n);

  hipMemcpy(matrix, d_matrix, n * n * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_matrix);
}

// test implementation

using vec = std::vector<int>;

// Overload the operator<< to print a std::vector<std::vector<int>>
std::ostream& operator<<(std::ostream& os, const vec& matrix) {
  std::size_t n = static_cast<std::size_t>(std::sqrt(matrix.size()));

  for (std::size_t row = 0; row < n; ++row) {
    for (std::size_t col = 0; col < n; ++col) {
      os << matrix[row * n + col] << ' ';
    }
    os << '\n';
  }
  return os;
}

#define ASSERT(expression)                                                     \
  if (!(expression)) {                                                         \
    std::cerr << "Assertion failed: " << #expression << "\nFile: " << __FILE__ \
              << "\n"                                                          \
              << "\nLine: " << __LINE__ << "\n";                               \
    return false;                                                              \
  }

#define ASSERT_EQUAL(a, b)                                          \
  if ((a) != (b)) {                                                 \
    std::cerr << "Assertion failed: " << #a << " == " << #b << "\n" \
              << #a << " = " << (a) << ", " << #b << " = " << (b)   \
              << "\nFile: " << __FILE__ << "\n"                     \
              << "\nLine: " << __LINE__ << "\n";                    \
    return false;                                                   \
  }

bool run_tests() {
  std::vector<std::tuple<vec, vec>> test_cases;
  test_cases.emplace_back(vec{}, vec{});
  test_cases.emplace_back(vec{1, 2, 3, 4}, vec{3, 1, 4, 2});
  test_cases.emplace_back(vec{1, 2, 3, 4, 5, 6, 7, 8, 9},
                          vec{7, 4, 1, 8, 5, 2, 9, 6, 3});
  test_cases.emplace_back(
      vec{1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16},
      vec{13, 9, 5, 1, 14, 10, 6, 2, 15, 11, 7, 3, 16, 12, 8, 4});
  test_cases.emplace_back(vec{1, 1, 1, 1}, vec{1, 1, 1, 1});

  for (auto& [input, expected] : test_cases) {
    int n = static_cast<int>(std::sqrt(input.size()));
    rotate_matrix(input.data(), n);
    ASSERT_EQUAL(input, expected);
  }

  return true;
}

int main() {
  if (run_tests()) {
    std::cout << "CUDA test succeeded\n";
    return 0;
  } else {
    std::cout << "Failure in CUDA tests\n";
    return 1;
  }
}
